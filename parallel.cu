//Parallel CUDA code to find parameters of gaussian distribution 
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include <bits/stdc++.h>

//N is the size of input array which we assume as points of gaussian distribution
#define N 10000000

using namespace std;

//This kernel finds the sum of the given input numbers in parallel by reducing the array recursively
__global__ void add(const float *input, float *output, int size) {

//Shared memory to store intermediate results among the threads
  __shared__ float partial_sum[256];
  
  int tid = threadIdx.x;
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  float sum = 0.0f;

  while (i < size) {
    sum += input[i];
    i += blockDim.x * gridDim.x;
  }
  
  partial_sum[tid] = sum;
  __syncthreads();

//Reducing the size of array and finding the sum of partial array recursively
  for (int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      partial_sum[tid] += partial_sum[tid + s];
    }
    __syncthreads();
  }

//Storing the result of current thread block in ouput
  if (tid == 0) {
    output[blockIdx.x] = partial_sum[0] ;
  }
}

//This kernel finds the square mean deviation sum of the given input numbers in parallel by reducing the array recursively
__global__ void square_add(const float *input, float *output, int size,float mean) {

//Shared memory among the threads
  __shared__ float partial_sum[256];
  int tid = threadIdx.x;
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  float sum = 0.0f;

  while (i < size) {
    sum += (input[i]-mean)*(input[i]-mean);
    i += blockDim.x * gridDim.x;
  }
  partial_sum[tid] = sum;
  __syncthreads();

//Doing Parallel reduction recursively
  for (int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      partial_sum[tid] += (partial_sum[tid + s]-mean)*(partial_sum[tid + s]-mean);
    }
    __syncthreads();
  }

//Storing result of current thread block
  if (tid == 0) {
    output[blockIdx.x] = partial_sum[0] ;
  }
}

int main(){
	srand(time(0));
	int blocks = N/256;
	//a contains input and ouput stores the intermediate values in finding mean,variance
	float *a,*output;
	float mean=0,variance=0,total=0,square_total=0;
	
	//Allocating memory for a,output
	hipMallocManaged(&a, N*sizeof(float));
	hipMallocManaged(&output, blocks*sizeof(float));
	
	//Assigning random numbers as input
	for(int i=0;i<N;i++)
	{
		a[i]= ((float)rand()) / RAND_MAX;
	}
	
	//Kernel launch for finding sum
	add<<<blocks, 256>>>(a, output, N);
	hipDeviceSynchronize();
	
	//Calculating mean
	for (int i = 0; i < blocks; i++)
    {
        total+= output[i];
    }
	mean=total/N;
	
	//Kernel launch for finding square sum
	square_add<<<blocks,256>>>(a,output,N,mean);
	hipDeviceSynchronize();
	
	//Calculating Variance
	for (int i = 0; i < blocks; i++)
    {
        square_total+= output[i];
    }
	variance=square_total/N;
	cout<<mean<<" "<<variance<<endl;
	hipFree(a);
	hipFree(output);
	return 0;
	}
